#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;
//typedefs and flash
int n,m;
ofstream fout;

struct point{
    int left;
    int right;
};

struct priorpoint{
    double fvalue;
    int left;
    int right;
    struct priorpoint* next;
};

struct node 
{ 
    int parent_row, parent_column; 
    double f, g, h; 
}; 


class compare{
    public:
    int operator() (const priorpoint& priorpoint1, const priorpoint& priorpoint2){
        return priorpoint1.fvalue > priorpoint2.fvalue;
    }
};



double doublemin(double d1,double d2){
    if(d1<d2){
        return d1;
    }
    return d2;
}
__device__ double doublemingpu(double d1,double d2){
    if(d1<d2){
        return d1;
    }
    return d2;
}
// A Utility Function to check whether given node (row, col) 
// is a Valid node or not. 
bool Valid(int row, int col,int *grid) 
{ 
    // Returns true if row number and column number 
    // is in range 
    if(row>=0&&row<n){
        if(col>=0&&col<m){
            if(grid[row*m+col]>=0){
                return true;
            }
        }
    } 
    return false;
} 

__device__ bool Validgpu(int row, int col,int *grid,int n,int m) 
{ 
    // Returns true if row number and column number 
    // is in range 
    if(row>=0&&row<n){
        if(col>=0&&col<m){
            if(grid[row*m+col]>=0){
                return true;
            }
        }
    } 
    return false;
} 

// A Utility Function to check whether destination node has 
// been reached or not 
bool Dest(int row, int col, point dest) 
{ 
    if (row == dest.left && col == dest.right){
        return true; 
    } 
    return false;
} 


// A Utility Function to calculate the 'h' heuristics. 
double Hestimate(int currow, int curcol, int destrow,int destcol) 
{ 
    int ll = currow-destrow;
    int rr = curcol-destcol;
    if(ll<0){
        ll=-ll;
    }
    if(rr<0){
        rr=-rr;
    }
    ll = ll + rr;
    double dd = ll;
    return dd;
} 

__device__ double Hestimategpu(int currow, int curcol, int destrow,int destcol) 
{ 
    int ll = currow-destrow;
    int rr = curcol-destcol;
    if(ll<0){
        ll=-ll;
    }
    if(rr<0){
        rr=-rr;
    }
    ll = ll + rr;
    double dd = ll;
    return dd;
} 


void path_trace(struct node** nodematrix, point dest, int hops){
    int row = dest.left;
    int col = dest.right;

    if((nodematrix[row][col].parent_row == row 
            && nodematrix[row][col].parent_column == col )){
        fout<<"The number of hops = "<<hops<<endl;
        fout<<"The path is =>"<<endl;
        fout<<"("<<row<<","<<col<<")";
        return;
    }
    int temp_row = nodematrix[row][col].parent_row; 
    int temp_col = nodematrix[row][col].parent_column; 
    int row2 = temp_row; 
    int col2 = temp_col;
    dest.left = row2;
    dest.right = col2;
    path_trace(nodematrix,dest,hops+1);
    fout<<"->";
    fout<<"("<<row<<","<<col<<")";
    if(hops==0){
        fout<<endl;
        fout<<"The cost of the optimal path is = "<<nodematrix[row][col].g<<endl;
    }
    return;
}

void path_trace2(struct node *nodematrix, point dest, int hops,int n,int m){
    int row = dest.left;
    int col = dest.right;

    if((nodematrix[row*m+col].parent_row == row 
            && nodematrix[row*m+col].parent_column == col )){
        fout<<"The number of hops = "<<hops<<endl;
        fout<<"The path is =>"<<endl;
        fout<<"("<<row<<","<<col<<")";
        return;
    }
    int temp_row = nodematrix[row*m+col].parent_row; 
    int temp_col = nodematrix[row*m+col].parent_column; 
    int row2 = temp_row; 
    int col2 = temp_col;
    dest.left = row2;
    dest.right = col2;
    path_trace2(nodematrix,dest,hops+1,n,m);
    fout<<"->";
    fout<<"("<<row<<","<<col<<")";
    if(hops==0){
        fout<<endl;
        fout<<"The cost of the optimal path is = "<<nodematrix[row*m+col].g<<endl;
    }
    return;
}

__device__ struct priorpoint* push1gpu(int i,int j,double f){
    struct priorpoint* temp;
    temp = (struct priorpoint*)malloc(sizeof(struct priorpoint));
    temp->fvalue = f;
    temp->left = i;
    temp->right = j;
    temp ->next = NULL;
    return temp;
}
__device__ struct priorpoint* push2gpu(int i,int j,double f,struct priorpoint* head){
    struct priorpoint* ptr;
    ptr = head;
    if(ptr->fvalue>f){
        struct priorpoint *temp;
        temp = (struct priorpoint*)malloc(sizeof(struct priorpoint));
        temp->fvalue = f;
        temp->left = i;
        temp->right = j;
        temp->next = head;
        return temp;
    }
    while((ptr->next !=NULL)&&((ptr->next)->fvalue<f)){
        ptr = ptr->next;
    }
    struct priorpoint *temp;
    temp = (struct priorpoint*)malloc(sizeof(struct priorpoint));
    temp->fvalue=f;
    temp->left = i;
    temp->right = j;
    if(ptr->next==NULL){
        temp->next = NULL;
        ptr->next = temp;
        return head;
    }
    else{
        temp->next = (ptr->next);
        ptr->next = temp;
        return head;
    }
}


struct priorpoint* push1cpu(int i,int j,double f){
    struct priorpoint* temp;
    temp = (struct priorpoint*)malloc(sizeof(struct priorpoint));
    temp->fvalue = f;
    temp->left = i;
    temp->right = j;
    temp ->next = NULL;
    return temp;
}

struct priorpoint* push2cpu(int i,int j,double f,struct priorpoint* head){
    struct priorpoint* ptr;
    ptr = head;
    if(ptr->fvalue>f){
        struct priorpoint *temp;
        temp = (struct priorpoint*)malloc(sizeof(struct priorpoint));
        temp->fvalue = f;
        temp->left = i;
        temp->right = j;
        temp->next = head;
        return temp;
    }
    while((ptr->next !=NULL)&&((ptr->next)->fvalue<f)){
        ptr = ptr->next;
    }
    struct priorpoint *temp;
    temp = (struct priorpoint*)malloc(sizeof(struct priorpoint));
    temp->fvalue=f;
    temp->left = i;
    temp->right = j;
    if(ptr->next==NULL){
        temp->next = NULL;
        ptr->next = temp;
        return head;
    }
    else{
        temp->next = (ptr->next);
        ptr->next = temp;
        return head;
    }
}

void astar(struct node** nodematrix,int *grid,struct point src,struct point dest){
    if (Valid (src.left, src.right,grid) == false || Valid (dest.left, dest.right,grid) == false) 
    { 
        printf ("Source or dest is inValid\n"); 
        return; 
    } 

    if (Dest(src.left, src.right, dest) == true) 
    { 
        printf ("We are already at the destination\n"); 
        return; 
    }
    bool visited[n][m]; 
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            visited[i][j]=false;
        }
    }

    int i, j; 

    i = src.left, j = src.right; 
    

    struct priorpoint* head;
    // struct priorpoint* tail;
    head = NULL;
    // tail = NULL;
    head = push1cpu(i,j,0);
    // tail = head;
    bool foundDest = false; 
    int count = 0;
    while (head != NULL) 
    { 
        count++;
        struct priorpoint  temp;
        temp.fvalue = head->fvalue;
        temp.left = head->left;
        temp.right = head->right;
        head = head->next;


        // Add this vertex to the closed list 
        i = temp.left; 
        j = temp.right; 
        visited[i][j] = true; 
        if(count == 20){
            //fout<<i<<" "<<j<<endl;
        }
        // To store the 'g', 'h' and 'f' of the 4 successors 
        double g2, h2, f2; 
        int vi, vj;
        int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
        int yy[8] = {0, 1, 0, -1,1,-1,1,-1};

        for(int pind = 0; pind<8; pind++){
            if(pind == 0){
           // printf("inside astarcpu\n");
        }
            
            vi = i + xx[pind];
            vj = j + yy[pind];
            
            if (Valid(vi, vj,grid) == true) 
            { 
                // If the destination node is the same as the current successor 
                if (Dest(vi, vj, dest) == true) 
                { 
                    // Set the Parent of the destination node 

                    nodematrix[vi][vj].parent_row = i; 
                    nodematrix[vi][vj].parent_column = j; 
                    g2 = nodematrix[i][j].g + grid[vi*m+vj]; 
                    h2 = Hestimate (vi, vj, dest.left, dest.right); 
                    f2 = g2 + h2;
                    nodematrix[vi][vj].f = f2; 
                    nodematrix[vi][vj].g = g2; 
                    nodematrix[vi][vj].h = h2;
                    foundDest = true; 
                    return; 
                } 
                // If the successor is nit visited
                if (!visited[vi][vj]) 
                { 
                    g2 = nodematrix[i][j].g + grid[vi*m+vj]; 
                    h2 = Hestimate (vi, vj, dest.left, dest.right); 
                    f2 = g2 + h2; 

                    // if the adjacent node is not in the minHeap insert it
                    //if it is present in the minHeap and the newer f is smaller than alreayy entered f than update
                    if(nodematrix[vi][vj].f > f2){
                        if(head == NULL){
                            head = push1cpu(vi,vj,f2);
                            // tail = head;
                        }else{
                            head = push2cpu(vi,vj,f2,head);
                        }
   
                        // Update the details of this node 
                        nodematrix[vi][vj].f = f2; 
                        nodematrix[vi][vj].g = g2; 
                        nodematrix[vi][vj].h = h2; 
                        nodematrix[vi][vj].parent_row = i; 
                        nodematrix[vi][vj].parent_column = j; 
                    }
                }
            }
        }
    }
    if(foundDest==false){
        //fout<<"Destination not found\n";
    }
}

void Nodematrixupdate(struct node** nodematrix,int *grid,int x,int y,point dest){
    
    bool visited[n][m]; 
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            visited[i][j]=false;
        }
    }
  
    int i, j; 

    i = x, j = y; 

    //initializing the added edge
    nodematrix[i][j].h = Hestimate(i,j,dest.left,dest.right);
    int XX[8] = {-1, 0, 1, 0,1,1,-1,-1};
    int YY[8] = {0, 1, 0, -1,1,-1,1,-1};
    double ming = nodematrix[i][j].g;
    int minindex=0;
    for(int pind=0;pind<8;pind++){
        int vi,vj;
        vi = i+XX[pind];
        vj = j+YY[pind];
        if(Valid(vi,vj,grid)){
            if(nodematrix[vi][vj].g<ming){
                ming = nodematrix[vi][vj].g;
                minindex = pind;
            }
        }
    }
    int parx,pary;
    parx = i+XX[minindex];
    pary = j+YY[minindex];
    nodematrix[i][j].g = ming+grid[i*m+j];
    nodematrix[i][j].f = ming+nodematrix[i][j].h;
    nodematrix[i][j].parent_row = parx;
    nodematrix[i][j].parent_column = pary;
    // finished initialisation
    //printf("%d %d\n",nodematrix[1][6].parent_row,nodematrix[1][6].parent_column);
    struct priorpoint* head;
    // struct priorpoint* tail;
    head = NULL;
    // tail = NULL;
    head = push1cpu(i,j,nodematrix[i][j].f);
    // tail = head;
    while (head != NULL) 
    { 
        struct priorpoint  temp;
        temp.fvalue = head->fvalue;
        temp.left = head->left;
        temp.right = head->right;
        head = head->next;


        // Add this vertex to the closed list 
        i = temp.left; 
        j = temp.right; 
        visited[i][j] = true; 
    
        // To store the 'g', 'h' and 'f' of the 4 successors 
        double g2, h2, f2; 
        int vi, vj;
        int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
        int yy[8] = {0, 1, 0, -1,1,-1,1,-1};

        for(int pind = 0; pind<8; pind++){
            
            vi = i + xx[pind];
            vj = j + yy[pind];
            
            if (Valid(vi, vj,grid) == true) 
            { 
                // If the destination node is the same as the current 
                // If the successor is not visited
                if (!visited[vi][vj]) 
                {   

                    g2 = doublemin(nodematrix[vi][vj].g,nodematrix[i][j].g + grid[vi*m+vj]); 
                    h2 = Hestimate (vi, vj, dest.left, dest.right); 
                    f2 = g2 + h2; 

                    // if the adjacent node is not in the minHeap insert it
                    //if it is present in the minHeap and the newer f is smaller than alreayy entered f than update
                    if(nodematrix[vi][vj].f > f2){
                        if(head == NULL){
                            head = push1cpu(vi,vj,f2);
                            // tail = head;
                        }else{
                            head = push2cpu(vi,vj,f2,head);
                        }
   
                        // Update the details of this node 
                        nodematrix[vi][vj].f = f2; 
                        nodematrix[vi][vj].g = g2; 
                        nodematrix[vi][vj].h = h2; 
                        nodematrix[vi][vj].parent_row = i; 
                        nodematrix[vi][vj].parent_column = j; 
                    }
                }
            }
        }
    }
}




__global__ void setvisited(bool* visited,int n,int m){
    int id = (blockIdx.x)*blockDim.x + threadIdx.x;
    if(id < n*m){
        visited[id] = false;
    }
}



__global__ void setpushpq(int* pushpq){
    int id = threadIdx.x;
    if(id < 8){
    pushpq[id] = 0;}
}


__global__ void astargpu(struct node *nodematrix, int *grid, int srcx,int srcy,int destx,int desty,int n,int m,bool* visited){
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            int id = i*m+j;
            visited[id] = false;
        }
    }  
    //printf("Hello-1\n");
    // for(int i=0;i<n;i++){
    //     for(int j=0;j<m;j++){
    //         visited[i*m+j]=false;
    //     }
    // }

    int i, j; 
    i = srcx, j = srcy; 
    
    struct priorpoint* head;
    
    head = NULL;
  
    head = push1gpu(i,j,0);
    //printf("Hello-2\n");
    bool foundDest = false; 
  
    int count = 0;
    int num1=0;
    while (head != NULL&&foundDest==false) 
    { 
        count++;
        struct priorpoint  temp;
        temp.fvalue = head->fvalue;
        temp.left = head->left;
        temp.right = head->right;
        head = head->next;
    //printf("Hello 3,1st loop\n");
        // Add this vertex to the closed list 

        i = temp.left; 
        j = temp.right; 
        visited[i*m+j] = true; 
       
        // To store the 'g', 'h' and 'f' of the 4 successors 
        //int vi, vj;
        int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
        int yy[8] = {0, 1, 0, -1,1,-1,1,-1};
        
        for(int itr=0;itr<8;itr++){

            int vi,vj;
            vi = i + xx[itr];
            vj = j + yy[itr];
            double f2,g2,h2;
    
            if(Validgpu(vi,vj,grid,n,m)){
                if(vi==destx&&vj==desty){
                    nodematrix[vi*m+vj].parent_row = i; 
                    nodematrix[vi*m+vj].parent_column = j; 
                    g2 = nodematrix[i*m+j].g + grid[vi*m+vj]; 
                    h2 = Hestimategpu (vi, vj, destx, desty); 
                    f2 = g2 + h2;
                    nodematrix[vi*m+vj].f = f2; 
                    nodematrix[vi*m+vj].g = g2; 
                    nodematrix[vi*m+vj].h = h2;
                    //pushpq[threadIdx.x]=2;
                    foundDest = true; 
                }
                if(!visited[vi*m+vj]){
                    g2 = nodematrix[i*m+j].g + grid[vi*m+vj]; 
                    h2 = Hestimategpu (vi, vj, destx, desty); 
                    f2 = g2 + h2;
                    if(nodematrix[vi*m+vj].f > f2){
                        if(head==NULL){
                            head = push1gpu(vi,vj,f2);
                        }
                        else{
                        //num1++;
                             head = push2gpu(vi,vj,f2,head);        
                        }
                        // Update the details of this node 
                        nodematrix[vi*m+vj].f = f2; 
                        nodematrix[vi*m+vj].g = g2; 
                        nodematrix[vi*m+vj].h = h2; 
                        nodematrix[vi*m+vj].parent_row = i; 
                        nodematrix[vi*m+vj].parent_column = j; 
                    }
                }
            }
        }
    }
}

__global__ void astargpuleveltwo(struct node *nodematrix, int *grid,int destx,int desty,int *pushpq, int n,int m,int i,int j,bool* visited){
    int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
    int yy[8] = {0, 1, 0, -1,1,-1,1,-1};
    int vi,vj;
    vi = i + xx[threadIdx.x];
    vj = j + yy[threadIdx.x];
    double f2,g2,h2;
    
    if(Validgpu(vi,vj,grid,n,m)){
        if(vi==destx&&vj==desty){
            nodematrix[vi*m+vj].parent_row = i; 
            nodematrix[vi*m+vj].parent_column = j; 
            g2 = nodematrix[i*m+j].g + grid[vi*m+vj]; 
            h2 = Hestimategpu (vi, vj, destx, desty); 
            f2 = g2 + h2;
            nodematrix[vi*m+vj].f = f2; 
            nodematrix[vi*m+vj].g = g2; 
            nodematrix[vi*m+vj].h = h2;
            pushpq[threadIdx.x]=2;
            //foundDest = true; 
        }
        if(!visited[vi*m+vj]){
            g2 = nodematrix[i*m+j].g + grid[vi*m+vj]; 
            h2 = Hestimategpu (vi, vj, destx, desty); 
            f2 = g2 + h2;
            if(nodematrix[vi*m+vj].f > f2){
                  pushpq[threadIdx.x] = 1;
                  // Update the details of this node 
                  nodematrix[vi*m+vj].f = f2; 
                  nodematrix[vi*m+vj].g = g2; 
                  nodematrix[vi*m+vj].h = h2; 
                  nodematrix[vi*m+vj].parent_row = i; 
                  nodematrix[vi*m+vj].parent_column = j; 
            }
        }
    }
}
__global__ void astargpulevelone(struct node *nodematrix, int *grid, int srcx,int srcy,int destx,int desty,int n,int m,bool* visited,int* pushpq){
    setvisited<<<n,m>>>(visited,n,m);
    hipDeviceSynchronize();
    //printf("Hello-1\n");
    // for(int i=0;i<n;i++){
    //     for(int j=0;j<m;j++){
    //         visited[i*m+j]=false;
    //     }
    // }

    int i, j; 
    i = srcx, j = srcy; 
    
    struct priorpoint* head;
    
    head = NULL;
  
    head = push1gpu(i,j,0);
    //printf("Hello-2\n");
    bool foundDest = false; 
  
    int count = 0;
    int num1=0;
    while (head != NULL&&foundDest==false) 
    { 
        count++;
        struct priorpoint  temp;
        temp.fvalue = head->fvalue;
        temp.left = head->left;
        temp.right = head->right;
        head = head->next;
    //printf("Hello 3,1st loop\n");
        // Add this vertex to the closed list 

        i = temp.left; 
        j = temp.right; 
        visited[i*m+j] = true; 
       
        // To store the 'g', 'h' and 'f' of the 4 successors 
        int vi, vj;
        int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
        int yy[8] = {0, 1, 0, -1,1,-1,1,-1};
        setpushpq<<<1,8>>>(pushpq);
        hipDeviceSynchronize();
        astargpuleveltwo<<<1,8>>>(nodematrix,grid,destx,desty,pushpq,n,m,i,j,visited);
        hipDeviceSynchronize();
        
        //check statement
        
        
        for(int itr=0;itr<8;itr++){
            vi = i+xx[itr];
            vj = j+yy[itr];
            if(pushpq[itr]==1){
                if(head==NULL){
                    head = push1gpu(vi,vj,nodematrix[vi*m+vj].f);
                }
                else{
                    num1++;
                    head = push2gpu(vi,vj,nodematrix[vi*m+vj].f,head);        
                }
            }
            else if(pushpq[itr]==2){
                foundDest=true;
            }
        }
    }
}

__global__ void Nodematrixupdategpu(struct node *nodematrix,int *grid,int x,int y,int e,int destx,int desty,int n,int m,bool* visited){
    grid[x*m+y] = e;
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            int id = i*m+j;
            visited[id] = false;
        }
    }  
    // for(int i=0;i<n;i++){
    //     for(int j=0;j<m;j++){
    //         visited[i*m+j]=false;
    //     }
    // }

    int i, j; 
    i = x, j = y; 
    
    nodematrix[i*m+j].h = Hestimategpu(i,j,destx,desty);
    int XX[8] = {-1, 0, 1, 0,1,1,-1,-1};
    int YY[8] = {0, 1, 0, -1,1,-1,1,-1};
    double ming = nodematrix[i*m+j].g;
    int minindex=0;
    for(int pind=0;pind<8;pind++){
        int vi,vj;
        vi = i+XX[pind];
        vj = j+YY[pind];
        if(Validgpu(vi,vj,grid,n,m)){
            if(nodematrix[vi*m+vj].g<ming){
                ming = nodematrix[vi*m+vj].g;
                minindex = pind;
            }
        }
    }
    int parx,pary;
    parx = i+XX[minindex];
    pary = j+YY[minindex];
    nodematrix[i*m+j].g = ming+grid[i*m+j];
    nodematrix[i*m+j].f = ming+nodematrix[i*m+j].h;
    nodematrix[i*m+j].parent_row = parx;
    nodematrix[i*m+j].parent_column = pary;
    //printf("%d %d\n",nodematrix[m+6].parent_row,nodematrix[m+6].parent_column);    
    struct priorpoint* head;
    // struct priorpoint* tail;
    head = NULL;
    // tail = NULL;
    head = push1gpu(i,j,nodematrix[i*m+j].f);
    // tail = head; 
    while (head != NULL) 
    { 
        struct priorpoint  temp;
        temp.fvalue = head->fvalue;
        temp.left = head->left;
        temp.right = head->right;
        head = head->next;
        // Add this vertex to the closed list 
        i = temp.left; 
        j = temp.right; 
        visited[i*m+j] = true; 
    
        // To store the 'g', 'h' and 'f' of the 4 successors 
        //int vi, vj;
        int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
        int yy[8] = {0, 1, 0, -1,1,-1,1,-1};

        hipDeviceSynchronize();
        //for(int pind = 0; pind<8; pind++){}
        for(int itr=0;itr<8;itr++){
            int vi,vj;
            double f2,g2,h2;
            vi = i + xx[itr];
            vj = j + yy[itr];
    
            if (Validgpu(vi, vj,grid,n,m) == true) 
            { 
                // If the destination node is the same as the current 
                // If the successor is not visited
                if (!visited[vi*m+vj]) 
                {   
                    g2 = doublemingpu(nodematrix[vi*m+vj].g,nodematrix[i*m+j].g + grid[vi*m+vj]); 
                    h2 = Hestimategpu (vi, vj, destx, desty); 
                    f2 = g2 + h2; 
                    // if the adjacent node is not in the minHeap insert it
                    //if it is present in the minHeap and the newer f is smaller than alreayy entered f than update
                    if(nodematrix[vi*m+vj].f > f2){
                    
                        if(head==NULL){
                            head = push1gpu(vi,vj,f2);
                        }
                        else{
                            head = push2gpu(vi,vj,f2,head);
                        }
                        // Update the details of this node 
                        nodematrix[vi*m+vj].f = f2; 
                        nodematrix[vi*m+vj].g = g2; 
                        nodematrix[vi*m+vj].h = h2; 
                        nodematrix[vi*m+vj].parent_row = i; 
                        nodematrix[vi*m+vj].parent_column = j; 
                    }
                }
            } 
        }
    }
}

__global__ void Nodematrixupdategpuleveltwo(struct node *nodematrix,int *grid,int destx,int desty,bool* visited,int i,int j,int n,int m,int *pushpq){
    int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
    int yy[8] = {0, 1, 0, -1,1,-1,1,-1};
    int vi,vj;
    double f2,g2,h2;
    vi = i + xx[threadIdx.x];
    vj = j + yy[threadIdx.x];
    
    if (Validgpu(vi, vj,grid,n,m) == true) 
            { 
                // If the destination node is the same as the current 
                // If the successor is not visited
                if (!visited[vi*m+vj]) 
                {   
                    g2 = doublemingpu(nodematrix[vi*m+vj].g,nodematrix[i*m+j].g + grid[vi*m+vj]); 
                    h2 = Hestimategpu (vi, vj, destx, desty); 
                    f2 = g2 + h2; 
                    // if the adjacent node is not in the minHeap insert it
                    //if it is present in the minHeap and the newer f is smaller than alreayy entered f than update
                    if(nodematrix[vi*m+vj].f > f2){
                    
                        pushpq[threadIdx.x] = 1;
                        // Update the details of this node 
                        nodematrix[vi*m+vj].f = f2; 
                        nodematrix[vi*m+vj].g = g2; 
                        nodematrix[vi*m+vj].h = h2; 
                        nodematrix[vi*m+vj].parent_row = i; 
                        nodematrix[vi*m+vj].parent_column = j; 
                    }
                }
            } 
}
__global__ void Nodematrixupdategpulevelone(struct node *nodematrix,int *grid,int x,int y,int e,int destx,int desty,int n,int m,bool* visited,int* pushpq){
    grid[x*m+y] = e;
    setvisited<<<n,m>>>(visited,n,m);
    hipDeviceSynchronize();
    // for(int i=0;i<n;i++){
    //     for(int j=0;j<m;j++){
    //         visited[i*m+j]=false;
    //     }
    // }

    int i, j; 
    i = x, j = y; 
    
    nodematrix[i*m+j].h = Hestimategpu(i,j,destx,desty);
    int XX[8] = {-1, 0, 1, 0,1,1,-1,-1};
    int YY[8] = {0, 1, 0, -1,1,-1,1,-1};
    double ming = nodematrix[i*m+j].g;
    int minindex=0;
    for(int pind=0;pind<8;pind++){
        int vi,vj;
        vi = i+XX[pind];
        vj = j+YY[pind];
        if(Validgpu(vi,vj,grid,n,m)){
            if(nodematrix[vi*m+vj].g<ming){
                ming = nodematrix[vi*m+vj].g;
                minindex = pind;
            }
        }
    }
    int parx,pary;
    parx = i+XX[minindex];
    pary = j+YY[minindex];
    nodematrix[i*m+j].g = ming+grid[i*m+j];
    nodematrix[i*m+j].f = ming+nodematrix[i*m+j].h;
    nodematrix[i*m+j].parent_row = parx;
    nodematrix[i*m+j].parent_column = pary;
    //printf("%d %d\n",nodematrix[m+6].parent_row,nodematrix[m+6].parent_column);    
    struct priorpoint* head;
    // struct priorpoint* tail;
    head = NULL;
    // tail = NULL;
    head = push1gpu(i,j,nodematrix[i*m+j].f);
    // tail = head; 
    while (head != NULL) 
    { 
        struct priorpoint  temp;
        temp.fvalue = head->fvalue;
        temp.left = head->left;
        temp.right = head->right;
        head = head->next;
        // Add this vertex to the closed list 
        i = temp.left; 
        j = temp.right; 
        visited[i*m+j] = true; 
    
        // To store the 'g', 'h' and 'f' of the 4 successors 
        int vi, vj;
        int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
        int yy[8] = {0, 1, 0, -1,1,-1,1,-1};
        setpushpq<<<1,8>>>(pushpq);
        hipDeviceSynchronize();
        //for(int pind = 0; pind<8; pind++){}
        Nodematrixupdategpuleveltwo<<<1,8>>>(nodematrix,grid,destx,desty,visited,i,j,n,m,pushpq);
        hipDeviceSynchronize();//check statement
        for(int itr=0;itr<8;itr++){
            vi = i+xx[itr];
            vj = j+yy[itr];
            if(pushpq[itr]==1){
                if(head==NULL){
                    head = push1gpu(vi,vj,nodematrix[vi*m+vj].f);
                }
                else{
                    head = push2gpu(vi,vj,nodematrix[vi*m+vj].f,head);
                }
            }
        }
    }
    //printf("%d %d\n",nodematrix[m+6].parent_row,nodematrix[m+6].parent_column);
}
__global__ void Nodematrixtrivial(struct node *nodematrix,int *grid,int x,int y,int e,int destx,int desty,int n,int m){
    nodematrix[x*m+y].h = Hestimategpu(x,y,destx,desty);
    grid[x*m+y] = e;
}

__global__ void initialize(struct node *nodematrix,int n,int m,int srcx,int srcy){
    int id = ((blockIdx.x)*m)+threadIdx.x;

    if(id<n*m){
        if(id == (srcx*m)+srcy){
            //printf("srid %d\n",id);
            nodematrix[id].f=0;
            nodematrix[id].g=0;
            nodematrix[id].h=0;
            nodematrix[id].parent_row=srcx;
            nodematrix[id].parent_column=srcy;
        }
        else{
            //printf("id %d\n",id);
            nodematrix[id].f=FLT_MAX;
            nodematrix[id].g=FLT_MAX;
            nodematrix[id].h=FLT_MAX;
            nodematrix[id].parent_row=-1;
            nodematrix[id].parent_column=-1;
        }

    }
}

__global__ void initialize2(struct node *nodematrix,int n,int m,int srcx,int srcy){
    for(int i1=0;i1<n;i1++){
        for(int j1=0;j1<m;j1++){
            int id = i1*m+j1;
            if(id == (srcx*m)+srcy){
                //printf("srid %d\n",id);
                nodematrix[id].f=0;
                nodematrix[id].g=0;
                nodematrix[id].h=0;
                nodematrix[id].parent_row=srcx;
                nodematrix[id].parent_column=srcy;
            }
            else{
                //printf("id %d\n",id);
                nodematrix[id].f=FLT_MAX;
                nodematrix[id].g=FLT_MAX;
                nodematrix[id].h=FLT_MAX;
                nodematrix[id].parent_row=-1;
                nodematrix[id].parent_column=-1;
            }
        }
    }
}


int main(){ 
    //event-1 start

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

  FILE *inputfilepointer;
    char *inputfilename = "input.txt";
    inputfilepointer    = fopen( inputfilename , "r");
    char *outputfilename = "output.txt";
        fout.open(outputfilename);
    //fout<<"outputfile ha s been opened"<<endl;
     //Checking if file ptr is NULL
     if ( inputfilepointer == NULL )  {
           printf( "input.txt file failed to open." );
               return 0;
     }
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of rows
    fscanf( inputfilepointer, "%d", &m );

    int *grid;
    grid = (int *)malloc(n*m*sizeof(int));
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            int temp;
            fscanf( inputfilepointer, "%d", &temp );
            grid[i*m+j] = temp;
        }   
    }

    int srcx,srcy,destx,desty,Q;
    fscanf( inputfilepointer, "%d", &srcx );
fscanf( inputfilepointer, "%d", &srcy );
fscanf( inputfilepointer, "%d", &destx );
fscanf( inputfilepointer, "%d", &desty );
fscanf( inputfilepointer, "%d", &Q );


    struct point src,dest;
    src.left = srcx;
    src.right= srcy;
    dest.left = destx;
    dest.right = desty; 

    
    struct node** nodematrix;
    nodematrix = (struct node**)malloc(n*sizeof(struct node*));

    int i, j; 
    for(i=0;i<n;i++){
        nodematrix[i] = (struct node*)malloc(m*sizeof(struct node));
    }
    
    for (i=0; i<n; i++) 
    { 
        for (j=0; j<m; j++) 
        { 
            nodematrix[i][j].f = FLT_MAX; 
            nodematrix[i][j].g = FLT_MAX; 
            nodematrix[i][j].h = FLT_MAX; 
            nodematrix[i][j].parent_row = -1; 
            nodematrix[i][j].parent_column = -1; 
        } 
    }

    i = src.left, j = src.right; 
    nodematrix[i][j].f = 0.0; 
    nodematrix[i][j].g = 0.0; 
    nodematrix[i][j].h = 0.0; 
    nodematrix[i][j].parent_row = i; 
    nodematrix[i][j].parent_column = j; 
    int depth = 0;
    fout<<"CPU OUTPUT"<<endl;
    for(int itr=0;itr<Q;itr++){
        int op;
    fscanf( inputfilepointer, "%d", &op );
        if(op==7){
            if(depth == 0){
                astar(nodematrix,grid, src, dest); 
                depth++;
            }
            if(nodematrix[destx][desty].g == FLT_MAX){
                fout<<"The destination node is not found"<<endl;
                continue;
            }
            fout<<"The destination node is found\n"; 
            path_trace (nodematrix, dest,0); 
            fout<<"---------------------------------------------\n";
            
            continue;
        }
        int edges;
        fscanf( inputfilepointer, "%d", &edges );
        for(int itr1=0;itr1<edges;itr1++){
            int x,y,e;
            fscanf( inputfilepointer, "%d", &x );
        fscanf( inputfilepointer, "%d", &y );
            fscanf( inputfilepointer, "%d", &e );
            grid[x*m+y] = e;
            int num =0;
            int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
            int yy[8] = {0, 1, 0, -1,1,-1,1,-1};
            for(int it=0;it<8;it++){
                int a,b;
                a= x + xx[it];
                b = y + yy[it];
                if(grid[a*m+b] != -1){
                    num++;
                }
            }
            if(num == 0){
                nodematrix[x][y].h = Hestimate(x,y,destx,desty);
                continue;
            }
            Nodematrixupdate(nodematrix,grid,x,y,dest);
        }
        fout<<"Edges added"<<endl;
        fout<<"---------------------------------------------"<<endl;
    }   
    //event-1 ended



    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by CPU to execute is: %.6f ms\n", milliseconds);

    //event-2 starts

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    float milliseconds1 = 0;
    hipEventRecord(start1,0);

    fout<<"GPU OUTPUT"<<endl;
    fclose(inputfilepointer);
    inputfilepointer    = fopen( inputfilename , "r");
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of rows
    fscanf( inputfilepointer, "%d", &m );
    int *cpugrid,*gpugrid;
    cpugrid = (int*)malloc(n*m*sizeof(int));
    hipMalloc(&gpugrid,n*m*sizeof(int));
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            fscanf( inputfilepointer, "%d", &cpugrid[i*m+j] );
        }   
      }
    hipMemcpy(gpugrid,cpugrid,n*m*sizeof(int),hipMemcpyHostToDevice);
    fscanf( inputfilepointer, "%d", &srcx );
    fscanf( inputfilepointer, "%d", &srcy );
    fscanf( inputfilepointer, "%d", &destx );
    fscanf( inputfilepointer, "%d", &desty );
    fscanf( inputfilepointer, "%d", &Q );
    src.left = srcx;
    src.right= srcy;
    dest.left = destx;
    dest.right = desty; 
    struct node *cpunodematrix,*gpunodematrix;
    cpunodematrix = (node*)malloc(n*m*sizeof(node));
    hipMalloc(&gpunodematrix,(n*m)*sizeof(node));
    //printf("hi");
    //bharath<<<1,1>>>();
    //hipDeviceSynchronize();
    //printf("gi");
    initialize<<<n,m>>>(gpunodematrix,n,m,srcx,srcy);
    hipMemcpy(cpunodematrix,gpunodematrix,n*m*sizeof(struct node),hipMemcpyDeviceToHost);
    bool* visited;
    hipMalloc(&visited,n*m*sizeof(bool));
    int* pushpq;
    hipMalloc(&pushpq,8*sizeof(int));
    //initialisation done
    depth=0;
    //printf("Q %d\n",Q);
    for(int itr=0;itr<Q;itr++){
    //fout<<"inside the main gpu loop"<<endl;
        int op;
          fscanf( inputfilepointer, "%d", &op );
          
        //fout<<op<<endl;
        if(op==7){
           // printf("op %d\n",op);
            if(depth == 0){
        //fout<<"hola1"<<endl;
                astargpulevelone<<<1,1>>>(gpunodematrix,gpugrid, srcx,srcy, destx,desty,n,m,visited,pushpq); 
                hipMemcpy(cpunodematrix,gpunodematrix,n*m*sizeof(struct node),hipMemcpyDeviceToHost);
                //fout<<"exited astarone"<<endl;
                // setpushpq<<<1,8>>>(pushpq);
                // hipDeviceSynchronize();
                // astargpuleveltwo<<<1,1>>>(gpunodematrix,gpugrid,destx,desty,pushpq,n,m,4,0,visited);
                
                //fout<<"hola2"<<endl;
                depth++;
            }
            if(cpunodematrix[destx*m+desty].g==FLT_MAX){
                fout<<"The destination node is not found"<<endl;
                continue;
            }
            fout<<"The destination node is found"<<endl; 
            path_trace2(cpunodematrix, dest,0,n,m); 
            fout<<"---------------------------------------------"<<endl;
            

            continue;
        }
    
        int edges;
        fscanf( inputfilepointer, "%d", &edges );
        for(int itr1=0;itr1<edges;itr1++){
            int x,y,e;
            fscanf( inputfilepointer, "%d", &x );
              fscanf( inputfilepointer, "%d", &y );
            fscanf( inputfilepointer, "%d", &e );
            cpugrid[x*m+y] = e; 
            int num =0;
            int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
            int yy[8] = {0, 1, 0, -1,1,-1,1,-1};
            for(int it=0;it<8;it++){
                int a,b;
                a= x + xx[it];
                b = y + yy[it];
                if(a<0||a>=n){
                    continue;
                }
                if(b<0||b>=m){
                    continue;
                }
                if(cpugrid[a*m+b] != -1){
                    num++;
                }
            }
            if(num == 0){
                Nodematrixtrivial<<<1,1>>>(gpunodematrix,gpugrid,x,y,e,destx,desty,n,m);
                hipDeviceSynchronize();
                continue;
            }
            Nodematrixupdategpulevelone<<<1,1>>>(gpunodematrix,gpugrid,x,y,e,destx,desty,n,m,visited,pushpq);
            hipDeviceSynchronize();
        }
        hipMemcpy(cpunodematrix,gpunodematrix,n*m*sizeof(node),hipMemcpyDeviceToHost);
        fout<<"Edges added"<<endl;
        fout<<"---------------------------------------------"<<endl;
    }
    //event2 -ended
hipError_t err = hipGetLastError();
      if ( err != hipSuccess )
   {
      printf("CUDA Error: %s\n", hipGetErrorString(err));
      return 0;
   }
    hipEventRecord(stop1,0);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&milliseconds1, start1, stop1);
    printf("Time taken by GPU to execute is: %.6f ms\n", milliseconds1);
    
    //event-2 ended
    //event-3 starts

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    float milliseconds2 = 0;
    hipEventRecord(start2,0);

    fout<<"GPU(sequential) OUTPUT"<<endl;
    fclose(inputfilepointer);
    inputfilepointer    = fopen( inputfilename , "r");
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of rows
    fscanf( inputfilepointer, "%d", &m );
    int *cpugrid2,*gpugrid2;
    cpugrid2 = (int*)malloc(n*m*sizeof(int));
    hipMalloc(&gpugrid2,n*m*sizeof(int));
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            fscanf( inputfilepointer, "%d", &cpugrid2[i*m+j] );
        }   
      }
    hipMemcpy(gpugrid2,cpugrid2,n*m*sizeof(int),hipMemcpyHostToDevice);
    fscanf( inputfilepointer, "%d", &srcx );
    fscanf( inputfilepointer, "%d", &srcy );
    fscanf( inputfilepointer, "%d", &destx );
    fscanf( inputfilepointer, "%d", &desty );
    fscanf( inputfilepointer, "%d", &Q );
    src.left = srcx;
    src.right= srcy;
    dest.left = destx;
    dest.right = desty; 
    struct node *cpunodematrix2,*gpunodematrix2;
    cpunodematrix2 = (node*)malloc(n*m*sizeof(node));
    hipMalloc(&gpunodematrix2,(n*m)*sizeof(node));
    //printf("hi");
    //bharath<<<1,1>>>();
    //hipDeviceSynchronize();
    //printf("gi");
    initialize2<<<1,1>>>(gpunodematrix2,n,m,srcx,srcy);
    hipMemcpy(cpunodematrix2,gpunodematrix2,n*m*sizeof(struct node),hipMemcpyDeviceToHost);
    bool* visited2;
    hipMalloc(&visited2,n*m*sizeof(bool));

    //initialisation done
    depth=0;
    //printf("Q %d\n",Q);
    for(int itr=0;itr<Q;itr++){
    //fout<<"inside the main gpu loop"<<endl;
        int op;
          fscanf( inputfilepointer, "%d", &op );
          
        //fout<<op<<endl;
        if(op==7){
           // printf("op %d\n",op);
            if(depth == 0){
        //fout<<"hola1"<<endl;
                astargpu<<<1,1>>>(gpunodematrix2,gpugrid2, srcx,srcy, destx,desty,n,m,visited2); 
                hipMemcpy(cpunodematrix2,gpunodematrix2,n*m*sizeof(struct node),hipMemcpyDeviceToHost);
                //fout<<"exited astarone"<<endl;
                // setpushpq<<<1,8>>>(pushpq);
                // hipDeviceSynchronize();
                // astargpuleveltwo<<<1,1>>>(gpunodematrix,gpugrid,destx,desty,pushpq,n,m,4,0,visited);
                
                //fout<<"hola2"<<endl;
                depth++;
            }
            if(cpunodematrix2[destx*m+desty].g==FLT_MAX){
                fout<<"The destination node is not found"<<endl;
                continue;
            }
            fout<<"The destination node is found"<<endl; 
            path_trace2(cpunodematrix2, dest,0,n,m); 
            fout<<"---------------------------------------------"<<endl;
            

            continue;
        }
    
        int edges;
        fscanf( inputfilepointer, "%d", &edges );
        for(int itr1=0;itr1<edges;itr1++){
            int x,y,e;
            fscanf( inputfilepointer, "%d", &x );
              fscanf( inputfilepointer, "%d", &y );
            fscanf( inputfilepointer, "%d", &e );
            cpugrid2[x*m+y] = e; 
            int num =0;
            int xx[8] = {-1, 0, 1, 0,1,1,-1,-1};
            int yy[8] = {0, 1, 0, -1,1,-1,1,-1};
            for(int it=0;it<8;it++){
                int a,b;
                a= x + xx[it];
                b = y + yy[it];
                if(a<0||a>=n){
                    continue;
                }
                if(b<0||b>=m){
                    continue;
                }
                if(cpugrid2[a*m+b] != -1){
                    num++;
                }
            }
            if(num == 0){
                Nodematrixtrivial<<<1,1>>>(gpunodematrix2,gpugrid2,x,y,e,destx,desty,n,m);
                hipDeviceSynchronize();
                continue;
            }
            Nodematrixupdategpu<<<1,1>>>(gpunodematrix2,gpugrid2,x,y,e,destx,desty,n,m,visited2);
            hipDeviceSynchronize();
        }
        hipMemcpy(cpunodematrix2,gpunodematrix2,n*m*sizeof(node),hipMemcpyDeviceToHost);
        fout<<"Edges added"<<endl;
        fout<<"---------------------------------------------"<<endl;
    }
    //event2 -ended
    hipError_t err2 = hipGetLastError();
      if ( err2 != hipSuccess )
   {
      printf("CUDA Error: %s\n", hipGetErrorString(err2));
      return 0;
   }
    hipEventRecord(stop2,0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&milliseconds2, start2, stop2);
    printf("Time taken by GPU(sequential) to execute is: %.6f ms\n", milliseconds2);

    return 0;
}
