#include <algorithm>
#include<bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

class Node{
public:
    
    int size=0;
    bool leaf;
    int keys[9];
    int values[8][20];
    Node* pointers[9];
    Node* next=NULL;//leaf node
    //Node* parent=NULL;

    Node(){
        size=0;
        leaf=false;
    }
};



Node* findpar(Node* child,Node* root){
    if(root->leaf==true||root->pointers[0]->leaf==true){
        return NULL;
    }
    for(int i=0;i<root->size+1;i++){
        if(root->pointers[i]==child){
            return root;
        }
        else if(findpar(child,root->pointers[i])!=NULL){
            return findpar(child,root->pointers[i]);
        }
    }
    return NULL;
}
Node* insertinternal(Node* parent,Node* newchild, Node* root,int val){
    //parent may or may not have overflow
    if(parent->size<7){
        int itr=0;
        while(val>parent->keys[itr]&&itr<parent->size){
            itr++;
        }
        for(int i= parent->size-1;i>=itr;i--){
            parent->keys[i+1] = parent->keys[i];
        }
        for(int i=parent->size;i>itr;i--){
            parent->pointers[i+1] = parent->pointers[i];
        }
        parent->pointers[itr+1] = newchild;
        parent->size++;
        parent->keys[itr] = val;
        return root;
    }    
    //parent has overflow,handle this and call its parent with a new child node
    vector <int> temparray ;
    for(int i=0;i<8;i++){
        temparray.push_back(parent->keys[i]);
    } 
    vector <Node*> temppointers ;
    for(int i=0;i<=8;i++){
        temppointers.push_back(parent->pointers[i]);
    }
    int itr = 0;

    while(val>temparray[itr]&&itr<7){
        itr++;
    }
    for(int i=6;i>=itr;i--){
        temparray[i+1] = temparray[i];
    }
    for(int i=7;i>itr;i--){
        temppointers[i+1] = temppointers[i];
    }

    temppointers[itr+1] = newchild;
    temparray[itr] = val;
    //define new node and distribute keys and links
    Node* internalnode = new Node();
    internalnode->leaf=false;
    internalnode->size=3;
    parent->size = 4;

    for(int i=0;i<parent->size;i++){
        parent->keys[i] = temparray[i];
    }
    for(int i=0;i<5;i++){
        parent->pointers[i] = temppointers[i];
    }
    int transfer = temparray[4];
    for(int i=0;i<3;i++){
        internalnode->keys[i] = temparray[5+i];
    }
    for(int i=0;i<4;i++){
        internalnode->pointers[i] = temppointers[5+i];
    }

    if(parent==root){
        Node* root2 = new Node();
        root2->keys[0] = transfer;
        //root2->pointers[0] = ne atomicAdd(&(ptr->values[i][attr-2]),upd);w Node();
        root2->pointers[0] = parent;
        //root2->pointers[1] = new Node();
        root2->pointers[1] = internalnode;
        root2->size=1;
        root2->leaf=false;
        return root2;
    }
    //write condition if it is root.
    return insertinternal(findpar(parent,root),internalnode,root,transfer);
   
}

Node* insert(Node* root,int val,int m){
    if (root==NULL){
        root = new Node();
        for(int i=0;i<8;i++){
            for(int j=0;j<m-1;j++){
                root->values[i][j] = 0;
            }
        }
        root->leaf=true;
        root->keys[0]=val;
        root->size=1;
        return root;
    }
    Node* ptr = root;
    Node* parent;
    while(ptr->leaf==false){
        parent = ptr;
        for(int i=0;i<ptr->size;i++){
            if(val<ptr->keys[i]){
                ptr = ptr->pointers[i];
                break;
            }
            if(i==ptr->size-1){
                ptr = ptr->pointers[i+1];
                break;
            }
        }
    }
    if(ptr->size<7){
        int i=0;
        while(val>ptr->keys[i]&&i<ptr->size){
            i++;
        }
        for(int itr=ptr->size-1;itr>=i;itr--){
            ptr->keys[itr+1] = ptr->keys[itr];
        }
        ptr->keys[i] = val;

        ptr->size++;
        return root;
    }
    
    //overflow condition
    Node* leaf2 = new Node();
    for(int i=0;i<8;i++){
        for(int j=0;j<m-1;j++){
            leaf2->values[i][j]=0;
        }
    }
    leaf2->leaf=true;
    vector <int> temparray;
    for(int i=0;i<8;i++){
        temparray.push_back(ptr->keys[i]);
    }
    int itr = 0;

    while(val>temparray[itr]&&itr<7){
        itr++;
    }
    for(int i=6;i>=itr;i--){
        temparray[i+1] = temparray[i];
       
    }

    temparray[itr] = val;
    //cout<<itr<<endl;
    ptr->size = 4;
    leaf2->size = 4;
    leaf2->next = ptr->next;
    ptr->next = leaf2;
    //after inserting x,there is possibility that keys of ptr may change
    //so we update keys of both ptr and leaf2
    for(int i=0;i<ptr->size;i++){
        ptr->keys[i] = temparray[i];
    }
    for(int i=0;i<leaf2->size;i++){
        leaf2->keys[i] = temparray[ptr->size+i];
        //cout<<leaf2->keys[i]<<" ";
    }
    //cout<<endl;
    //updates done
    //now overflow mightve happened at root or internal node/leaf
    if(ptr==root){
        
        Node* root2 = new Node();
        
        root2->keys[0] = leaf2->keys[0];
        //root2->pointers[0] = new Node();
        root2->pointers[0] = ptr;
        //root2->pointers[1] = new Node();
        root2->pointers[1] = leaf2;
        root2->size=1;
        root2->leaf=false;
        return root2;
    }
    //overflow happened at some leaf node which is not the root.
    return insertinternal(parent,leaf2,root,leaf2->keys[0]);
}
bool searchtree(Node* root,int val){
    if(root==NULL){
        return false;
    }
    Node* ptr = root;
    while(ptr->leaf==false){
        for(int i=0;i<ptr->size;i++){
            if(val<ptr->keys[i]){
                ptr = ptr->pointers[i];
                break;
            }
            if(i==ptr->size-1){
                ptr=ptr->pointers[i+1];
                break;
            }
        }
    }
    for(int i=0;i<ptr->size;i++){
        if(ptr->keys[i]==val){
            cout<<ptr->keys[i]<<" ";
            for(int j=0;j<ptr->size;j++){
                cout<<ptr->values[i][j]<<" ";
            }
            cout<<endl;
            return true;
        }
    }
    return false;
}

__global__ void gpusearch(int n,int m,int num,int *gpuarray,int *gpuoutput, Node* gputree){
    //printf("hi");
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < num)
    {
    int val = gpuarray[id];
    Node* ptr = gputree;
    for(int i=0;i<m;i++){
        gpuoutput[id*m+i]=-1;
    }
    //printf("%d %d",ptr->keys[0],ptr->keys[1]);
    while(ptr->leaf==false){
        for(int i=0;i<ptr->size;i++){
            if(val<ptr->keys[i]){
                ptr = ptr->pointers[i];
                break;
            }
            if(i==ptr->size-1){
                ptr=ptr->pointers[i+1];
                break;
            }
        }
    }
    for(int i=0;i<ptr->size;i++){
        if(ptr->keys[i]==val){
            gpuoutput[id*m] = val;
            //printf("%d ",val);
            for(int j=0;j<m-1;j++){
                gpuoutput[id*m+1+j] = ptr->values[i][j];
                //printf("%d ",ptr->values[i][j]);
            }
            //printf("\n");
            return;
        }
    }
    //printf("-1\n");
    gpuoutput[id*m]=-1;
    }
    
}

__global__ void gpurange(int n,int m,int num,int *gpuarray,Node* gputree,Node **startpoint){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id<num)
    {
    int a = gpuarray[2*id];
    int b = gpuarray[2*id+1];
    int itr=0;
    Node* ptr=gputree;
    if(ptr==NULL){
        return;
    }
    while(ptr->leaf==false){
        for(int i=0;i<ptr->size;i++){
            if(a<ptr->keys[i]){
                ptr = ptr->pointers[i];
                break;
            }
            if(i==ptr->size-1){
                ptr=ptr->pointers[i+1];
                break;
            }
        }
    }
    startpoint[id] = ptr->next;
    }
}

__global__  void gpuadd(int n,int m,int num,int *gpuarray,Node* gputree,Node **leafpoint){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < num)
    {
    int key = gpuarray[id*3];
    int attr = gpuarray[id*3+1];
    int upd = gpuarray[id*3+2];
    Node* ptr = gputree;
    while(ptr->leaf==false){
        for(int i=0;i<ptr->size;i++){
            if(key<ptr->keys[i]){
                ptr = ptr->pointers[i];
                break;
            }
            if(i==ptr->size-1){
                ptr=ptr->pointers[i+1];
                break;
            }
        }
    }
    for(int i=0;i<ptr->size;i++){
        if(ptr->keys[i]==key){
            //ptr->values[m*i+attr-2] = upd;
            atomicAdd(&(ptr->values[i][attr-2]),upd);
        }
    }
     leafpoint[id] = ptr->next;
    }
}

__global__ void gpupaths(int n,int m,int val,int *gpuoutput,Node* gputree){
    int itr=0;
    Node* ptr = gputree;
    for(int i=0;i<n;i++){
        gpuoutput[i]=-1;
    }
    gpuoutput[itr]=ptr->keys[0];
    itr++;
    while(ptr->leaf==false){
        for(int i=0;i<ptr->size;i++){
            if(val<ptr->keys[i]){
                ptr = ptr->pointers[i];
                gpuoutput[itr]=ptr->keys[0];
                itr++;
                //v.push_back(ptr->keys[0]);
                break;
            }
            if(i==ptr->size-1){
                ptr=ptr->pointers[i+1];
                gpuoutput[itr]=ptr->keys[0];
                itr++;
                //v.push_back(ptr->keys[0]);
                break;
            }
        }
    }
    //v.push_bac
}


Node* gpucopy(Node* ptr){
    Node* temp;
    hipMalloc(&temp,sizeof(Node));
    hipMemcpy(temp,ptr,sizeof(Node),hipMemcpyHostToDevice);
    return temp;
}

Node* copy(Node* head,int m){
    Node* temp = new Node();
    temp->leaf = head->leaf;
    temp->size=head->size;
    for(int i=0;i<8;i++){
        temp->keys[i] = head->keys[i];
    }
    //temp->values=head->values;
    if(head->leaf==true){
        for(int i=0;i<8;i++){
            for(int j=0;j<m-1;j++){
                temp->values[i][j] = head->values[i][j];
            }
        }
        temp->next = head;
        return gpucopy(temp);
    }
    for(int i=0;i<=head->size;i++){   
        temp->pointers[i] = (copy(head->pointers[i],m));
    }
    return gpucopy(temp);
}


void main2 ( int n, int m, int q, int *database, int **queries ,char* outputfilename)  {
    ofstream fout;
    fout.open(outputfilename);
    Node* tree = NULL;
    for(int i=0;i<n;i++){
        tree = insert(tree,database[i*m],m);
    }
    //cout<<tree->keys[0]<<" "<<tree->keys[1]<<endl;
    Node* gputree = copy(tree,m);
    //copy tree into gputree.
    for(int i=0;i<q;i++){
        if(queries[i][0]==1){
            //cout<<"main2";
            int num = queries[i][1];
            int *gpuarray,*gpuoutput,*array;
            array = (int *) malloc(num*sizeof(int));
            hipMalloc(&gpuoutput,(num*m)*(sizeof(int)));
            hipMalloc(&gpuarray,num*(sizeof(int)));
            for(int j=0;j<num;j++){
                //cout<<queries[i][2+j]<<" ";
            }
            //cout<<endl;
            for(int j=0;j<num;j++){
                
                array[j] = queries[i][2+j];
                //cout<<queries[i][2+j]<<" ";
                //bool temp = searchtree(tree,queries[i][2+j]);
            }
           // cout<<endl;
            hipMemcpy(gpuarray,array,num*sizeof(int),hipMemcpyHostToDevice);
            gpusearch<<<11,num/10 + 1>>>(n,m,num,gpuarray,gpuoutput,gputree);
            hipDeviceSynchronize();
            int *output;
            output = (int *) malloc((num*m)*sizeof(int));
            hipMemcpy(output,gpuoutput,(num*m)*sizeof(int),hipMemcpyDeviceToHost);
            for(int j=0;j<num;j++){
                //cout<<"a";
                if(output[j*m]==-1){
                    fout<<"-1"<<endl;
                    continue;
                }
                for(int k=0;k<m;k++){
                    fout<<output[j*m+k]<<" ";
                }
                fout<<endl;
            }
        }
        else if(queries[i][0]==2){
            //continue;
            int num = queries[i][1];
            int *gpuarray,*array;
            array = (int *) malloc((2*num)*sizeof(int));
            hipMalloc(&gpuarray,(2*num)*sizeof(int));
            for(int j=0;j<2*num;j++){
                array[j] = queries[i][2+j];
            }
            hipMemcpy(gpuarray,array,(2*num)*sizeof(int),hipMemcpyHostToDevice);
            Node **startpoint;
            hipMalloc(&startpoint,num*sizeof(Node*));
            Node **cpustartpoint;
            gpurange<<<11,num/10 + 1>>>(n,m,num,gpuarray,gputree,startpoint);
            cpustartpoint = (Node**)malloc(num*sizeof(Node*));
            hipMemcpy(cpustartpoint,startpoint,num*sizeof(Node*),hipMemcpyDeviceToHost);
            for(int j=0;j<num;j++){
                Node* ptr = cpustartpoint[j];
                int a = array[2*j];
                int b = array[2*j+1];
                int itr=0;
                while(ptr!=NULL){
                    for(int k=0;k<ptr->size;k++){
                        if(ptr->keys[k]>=a&&ptr->keys[k]<=b){
                            itr++;
                            fout<<ptr->keys[k]<<" ";
                            for(int l=0;l<m-1;l++){
                                fout<<ptr->values[k][l]<<" ";
                            }
                            fout<<endl;
                        }
                    }
                    ptr=ptr->next;
                }
                if(itr==0){
                    fout<<"-1"<<endl;
                }
            }
        }
        else if(queries[i][0]==3){
            //no output reqd
            int num = queries[i][1];
            int *gpuarray,*array;
            Node **leafpoint,**cpuleafpoint;
            hipMalloc(&leafpoint,num*sizeof(Node*));
            cpuleafpoint = (Node**) malloc(num*sizeof(Node *));
            array = (int *) malloc((3*num)*sizeof(int));
            hipMalloc(&gpuarray,(3*num)*sizeof(int));
            for(int j=0;j<3*num;j++){
                array[j] = queries[i][2+j];
            }
            hipMemcpy(gpuarray,array,(3*num)*sizeof(int),hipMemcpyHostToDevice);
            gpuadd<<<11,num/10 + 1>>>(n,m,num,gpuarray,gputree,leafpoint);
            hipMemcpy(cpuleafpoint,leafpoint,num*sizeof(Node*),hipMemcpyDeviceToHost);
            for(int j=0;j<num;j++){
                Node* ptr= cpuleafpoint[j];
                int key,attr,upd;
                key = array[3*j];
                attr = array[3*j+1];
                upd = array[3*j+2];
                for(int k=0;k<ptr->size;k++){
                     if(ptr->keys[k]==key){
                         ptr->values[k][attr-2] += upd;
                     }                        
                }
            }
        }
        else{
            int *gpuoutput;
            int val = queries[i][1];
            hipMalloc(&gpuoutput,(n)*sizeof(int)); 
            gpupaths<<<1,1>>>(n,m,val,gpuoutput,gputree);
            int *output;
            output = (int *) malloc(n*sizeof(int));
            hipMemcpy(output,gpuoutput,n*sizeof(int),hipMemcpyDeviceToHost);
            for(int i=0;i<n;i++){
                if(output[i]==-1){
                    break;
                }
                fout<<output[i]<<" ";
            }
            fout<<endl;
        }
    }
}

int main(int argc,char **argv){

    //variable declarations
    int n,m,q;
    
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0;
    }
    
    
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of rows
    fscanf( inputfilepointer, "%d", &m );      //scaning for number of columns

    int *database = (int *) malloc(n*m*sizeof(int));
    for(int i=0;i<n;i++){
        for(int j=0;j<m;j++){
            fscanf( inputfilepointer, "%d", &database[i*m+j] );
        }
    }
    
    fscanf( inputfilepointer, "%d", &q );      //scanning for number of queries
    int **queries = (int **) malloc(q*sizeof(int *));
    for(int i=0;i<q;i++){
        int typeop;
        fscanf( inputfilepointer, "%d", &typeop );
        if(typeop==4){
            //cout<<"a"<<endl;
            queries[i] = (int *) malloc (2*sizeof(int));
            queries[i][0] = 4;
            fscanf( inputfilepointer, "%d", &queries[i][1]);
        }
        else if(typeop==3){
            //cout<<"b"<<endl;
            int num;
            fscanf( inputfilepointer, "%d", &num );

            queries[i] = (int *) malloc((2+3*num)*sizeof(int));
            queries[i][0] = 3;
            queries[i][1] = num;
            for(int j=0;j<3*num;j++){
                fscanf( inputfilepointer, "%d", &queries[i][2+j] );
            }
        }
        else if(typeop==2){
            //cout<<"c"<<endl;
            int num;
            fscanf( inputfilepointer, "%d", &num );
            queries[i] = (int *) malloc((2+2*num)*sizeof(int));
            queries[i][0] = 2;
            queries[i][1] = num;
            for(int j=0;j<2*num;j++){
                fscanf( inputfilepointer, "%d", &queries[i][2+j] );
            }
        }
        else {
            //cout<<"d"<<endl;
            int num;
            fscanf( inputfilepointer, "%d", &num );
            queries[i] = (int *) malloc((2+num)*sizeof(int));
            queries[i][0] = 1;
            queries[i][1] = num;
            for(int j=0;j<num;j++){
                fscanf( inputfilepointer, "%d", &queries[i][2+j] );
            }
        }
    }
    

    char *outputfilename = argv[2]; 
    
    main2 ( n, m, q, database, queries, outputfilename);
    //cout<<"done";
    fclose( inputfilepointer );
    return 0;
}
